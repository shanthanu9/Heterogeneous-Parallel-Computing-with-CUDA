
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
using namespace std;

#define BLOCK_SIZE 1024
#define CEIL(a, b) ((a-1)/b +1)

__global__ void vertex_parallel_bfs(int *depth, int *R, int *C, int n) {

	int id = threadIdx.x;

	for(int i = id; i < n; i+=blockDim.x) {
		depth[i] = 1e9;
	}

	__shared__ int current_depth;
	__shared__ int done;

	if(id == 0) {
		depth[id] = 0;
		current_depth = 0;
		done = false;
	}

	__syncthreads();

	while(!done) {
		if(id == 0)
			done = true;
		
		__syncthreads();

		for(int v = id; v < n; v+=blockDim.x) {
			
			if(depth[v] == current_depth) {

				done = false;
				for(int j = R[v]; j < R[v+1]; j++) {	
					int u = C[j];
					if(depth[u] > depth[v]+1) {
						depth[u] = depth[v]+1;
					}
				}
			}
		}

		if(id == 0)
			current_depth++;

		__syncthreads();
	}
}

int main(int argc, char *argv[]) {
	if(argc < 3) {
		cout<<"Expecting a file as command line arguement...";
		return 0;
	}
	
	freopen(argv[1], "r", stdin);

	int n,m;
	cin>>n>>m;

	int *h_R = (int*)malloc((n+1)*sizeof(int));

	for(int i = 0; i <= n; i++) {
		cin>>h_R[i];
	}

	int *h_C = (int*)malloc(h_R[n]*sizeof(int));

	for(int i = 0; i < h_R[n]; i++) {
		cin>>h_C[i];
	} 

	int *d_R, *d_C, *d_depth;

	hipMalloc((void**) &d_R, (n+1)*sizeof(int));
	hipMalloc((void**) &d_C, h_R[n]*sizeof(int));
	hipMalloc((void**) &d_depth, n*sizeof(int));

	hipMemcpy(d_R, h_R, (n+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, h_R[n]*sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	vertex_parallel_bfs<<<1, BLOCK_SIZE>>>(d_depth, d_R, d_C, n);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout<<"Compute time in GPU: "<<milliseconds<<"ms"<<endl;

	int *h_depth = (int*) malloc(n*sizeof(int));

	hipMemcpy(h_depth, d_depth, n*sizeof(int), hipMemcpyDeviceToHost);

	int *h_check_depth = (int*)malloc(n*sizeof(int));

	freopen(argv[2], "r", stdin);

	for(int i = 0; i < n; i++) {
		cin>>h_check_depth[i];
	}

	bool flag = true;
	int count = 0;

	for(int i = 0; i < n; i++) {
		if(h_depth[i] != h_check_depth[i]) {
			flag = false;
			count++;
		}
	}

	if(flag) {
		cout<<"Solution is correct!";
	}
	else {
		cout<<"Solution is incorrect!"<<endl;
		cout<<count<<" testcases failed.";
	}

	return 0;
}