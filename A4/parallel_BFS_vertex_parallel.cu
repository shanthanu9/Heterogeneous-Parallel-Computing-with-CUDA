
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
using namespace std;

#define BLOCK_SIZE 1024
#define CEIL(a, b) ((a-1)/b +1)

__global__ void vertex_parallel_bfs(int *d, int *R, int *C, int n, int *depth) {

	int id = threadIdx.x;

	for(int i = id; i < n; i+=blockDim.x) {
		d[i] = 1e9;
	}

	__shared__ int current_depth;
	__shared__ int done;

	if(id == 0) {
		d[id] = 0;
		current_depth = 0;
		done = false;
	}

	__syncthreads();

	while(!done) {
		if(id == 0)
			done = true;
		
		__syncthreads();

		for(int v = id; v < n; v+=blockDim.x) {
			
			if(d[v] == current_depth) {

				done = false;
				for(int j = R[v]; j < R[v+1]; j++) {	
					int u = C[j];
					if(d[u] == int(1e9)) {
						d[u] = d[v]+1;
					}
				}
			}
		}

		if(id == 0)
			current_depth++;

		__syncthreads();
	}

	if(id == 0)
		*depth = current_depth;

}

int main(int argc, char *argv[]) {
	if(argc < 3) {
		cout<<"Expecting a file as command line arguement...";
		return 0;
	}
	
	freopen(argv[1], "r", stdin);

	int n,m;
	cin>>n>>m;

	int *h_R = (int*)malloc((n+1)*sizeof(int));

	for(int i = 0; i <= n; i++) {
		cin>>h_R[i];
	}

	int *h_C = (int*)malloc(h_R[n]*sizeof(int));

	for(int i = 0; i < h_R[n]; i++) {
		cin>>h_C[i];
	} 

	int *d_R, *d_C, *d_d, *d_depth;

	hipMalloc((void**) &d_R, (n+1)*sizeof(int));
	hipMalloc((void**) &d_C, h_R[n]*sizeof(int));
	hipMalloc((void**) &d_d, n*sizeof(int));
	hipMalloc((void**) &d_depth, sizeof(int));

	hipMemcpy(d_R, h_R, (n+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, h_R[n]*sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	vertex_parallel_bfs<<<1, BLOCK_SIZE>>>(d_d, d_R, d_C, n, d_depth);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout<<"Compute time in GPU: "<<milliseconds<<"ms"<<endl;

	int *h_d = (int*) malloc(n*sizeof(int));
	int *h_depth = (int*) malloc(sizeof(int));

	hipMemcpy(h_d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_depth, d_depth, sizeof(int), hipMemcpyDeviceToHost);

	int *h_check_d = (int*)malloc(n*sizeof(int));

	freopen(argv[2], "r", stdin);

	for(int i = 0; i < n; i++) {
		cin>>h_check_d[i];
	}

	bool flag = true;
	int count = 0;
	const int errcount = 20;

	for(int i = 0; i < n; i++) {
		if(h_d[i] != h_check_d[i]) {
			flag = false;
			if(count < errcount) {
				cout<<i<<" : "<<h_d[i]<<" "<<h_check_d[i]<<endl; 
			}
			count++;
		}
	}

	if(flag) {
		cout<<"Solution is correct!"<<endl;
		cout<<"The depth of the given graph from node 0 is "<<(*h_depth)<<endl;
	}
	else {
		cout<<"Solution is incorrect!"<<endl;
		cout<<count<<" testcases failed."<<endl;
	}

	return 0;
}