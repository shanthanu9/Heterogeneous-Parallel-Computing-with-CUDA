
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
using namespace std;

#define BLOCK_SIZE 1024
#define CEIL(a, b) ((a-1)/b +1)

__global__ void work_efficient_parallel_bfs(int *d, int *R, int *C, int n, int *Q0, int *Q1, int *depth) {

	int id = threadIdx.x;

	for(int i = id; i < n; i+=blockDim.x) {
		d[i] = 1e9;
	}

	__shared__ int Q0_len, Q1_len, current_depth;

	if(id == 0) {
		d[id] = 0;
		Q0[id] = 0;
		Q0_len = 1;
		Q1_len = 0;
		current_depth = 0;
	}	
	
	__syncthreads();

	while(Q0_len) {
		for(int i = id; i < Q0_len; i+=blockDim.x) {
			int v = Q0[i];
			for(int j = R[v]; j < R[v+1]; j++) {
				int u = C[j];
				if(atomicCAS(&d[u], int(1e9), d[v]+1) == int(1e9)) {
					int t = atomicAdd(&Q1_len, 1);
					Q1[t] = u;
				}
			}
		}

		__syncthreads();

		for(int i = id; i < Q1_len; i+=blockDim.x) {
			Q0[i] = Q1[i];
		}

		__syncthreads();

		if(id == 0) {
			Q0_len = Q1_len;
			Q1_len = 0;
			current_depth++;
		}

		__syncthreads();
 	}

 	if(id == 0)
 		*depth = current_depth -1;
}


int main(int argc, char *argv[]) {
	if(argc < 3) {
		cout<<"Expecting a file as command line arguement...";
		return 0;
	}
	
	freopen(argv[1], "r", stdin);

	int n,m;
	cin>>n>>m;

	int *h_R = (int*)malloc((n+1)*sizeof(int));

	for(int i = 0; i <= n; i++) {
		cin>>h_R[i];
	}

	int *h_C = (int*)malloc(h_R[n]*sizeof(int));

	for(int i = 0; i < h_R[n]; i++) {
		cin>>h_C[i];
	} 

	int *d_R, *d_C, *d_d, *Q0, *Q1, *d_depth;

	hipMalloc((void**) &d_R, (n+1)*sizeof(int));
	hipMalloc((void**) &d_C, h_R[n]*sizeof(int));
	hipMalloc((void**) &d_d, n*sizeof(int));
	hipMalloc((void**) &Q0, n*sizeof(int));
	hipMalloc((void**) &Q1, n*sizeof(int));
	hipMalloc((void**) &d_depth, sizeof(int));

	hipMemcpy(d_R, h_R, (n+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, h_R[n]*sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	work_efficient_parallel_bfs<<<1, BLOCK_SIZE>>>(d_d, d_R, d_C, n, Q0, Q1, d_depth);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout<<"Compute time in GPU: "<<milliseconds<<"ms"<<endl;

	int *h_d = (int*) malloc(n*sizeof(int));
	int *h_depth = (int*) malloc(sizeof(int));

	hipMemcpy(h_d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_depth, d_depth, sizeof(int), hipMemcpyDeviceToHost);

	int *h_check_d = (int*)malloc(n*sizeof(int));

	freopen(argv[2], "r", stdin);

	for(int i = 0; i < n; i++) {
		cin>>h_check_d[i];
	}

	bool flag = true;
	int count = 0;
	const int errcount = 20;

	for(int i = 0; i < n; i++) {
		if(h_d[i] != h_check_d[i]) {
			flag = false;
			if(count < errcount) {
				cout<<i<<" : "<<h_d[i]<<" "<<h_check_d[i]<<endl; 
			}
			count++;
		}
	}

	if(flag) {
		cout<<"Solution is correct!"<<endl;
		cout<<"The depth of the given graph from node 0 is "<<(*h_depth)<<endl;
	}
	else {
		cout<<"Solution is incorrect!"<<endl;
		cout<<count<<" testcases failed."<<endl;
	}

	return 0;
}