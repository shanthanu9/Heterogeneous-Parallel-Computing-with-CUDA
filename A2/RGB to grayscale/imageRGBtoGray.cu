#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "wb.h"
using namespace std;

#define CEIL(a, b) ((a-1)/b +1)
#define BLOCK_SIZE 1024

__global__ void compute(float * deviceOutputImageData, float * deviceInputImageData) {
	int Id = blockDim.x * blockIdx.x + threadIdx.x;
	float r = deviceInputImageData[3*Id];
	float g = deviceInputImageData[3*Id+1];
	float b = deviceInputImageData[3*Id+2];
	deviceOutputImageData[Id] = (0.21f * r + 0.71f * g + 0.07f * b);
} 

int main(int argc, char *argv[]) {

	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;

	/* parse the input arguments */
	wbArg_t args = wbArg_read(argc, argv);

	inputImageFile = wbArg_getInputFile(args, 0);
	inputImage = wbImport(inputImageFile);

	imageWidth  = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage); // For this lab the value is always 3

	// Since the image is monochromatic, it only contains one channel
	outputImage = wbImage_new(imageWidth, imageHeight, 1);

	hostInputImageData  = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	hipMalloc((void **)&deviceInputImageData,
		imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceOutputImageData,
		imageWidth * imageHeight * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	hipMemcpy(deviceInputImageData, hostInputImageData,
		imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	wbTime_stop(Copy, "Copying data to the GPU");

	///////////////////////////////////////////////////////
	wbTime_start(Compute, "Doing the computation on the GPU");

	compute<<<CEIL(imageWidth*imageHeight, BLOCK_SIZE), BLOCK_SIZE>>>(deviceOutputImageData, deviceInputImageData);

	wbTime_stop(Compute, "Doing the computation on the GPU");

	///////////////////////////////////////////////////////
	wbTime_start(Copy, "Copying data from the GPU");
	hipMemcpy(hostOutputImageData, deviceOutputImageData,
		imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(args, outputImage);

	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
