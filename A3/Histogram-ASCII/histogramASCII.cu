#include "hip/hip_runtime.h"
#include "wb.h"
#include <bits/stdc++.h>

using namespace std;

#define BLOCK_SIZE 128
#define NUM_BINS 128
#define CEIL(a, b) ((a-1)/b +1)

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

__global__ void compute(unsigned int * deviceInput, unsigned int * deviceBins, unsigned int inputLength) {
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i >= inputLength)
		return;

	unsigned int index = deviceInput[i];

	__shared__ unsigned int m[NUM_BINS];

	m[threadIdx.x] = 0;
	
	__syncthreads();

	atomicAdd(&m[index], 1);

	__syncthreads();

	if(threadIdx.x < NUM_BINS)
		atomicAdd(&deviceBins[threadIdx.x], m[threadIdx.x]);

}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);

		if (abort)
			exit(code);
	}
}


int main(int argc, char *argv[]) {
	
	wbArg_t args;
	int inputLength;
	unsigned int *hostInput;
	unsigned int *hostBins;
	unsigned int *deviceInput;
	unsigned int *deviceBins;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int *)wbImportChar(wbArg_getInputFile(args, 0), &inputLength);
	hostBins = (unsigned int *)calloc(NUM_BINS, sizeof(unsigned int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating GPU memory.");
	// Allocating GPU memory
	hipMalloc((void **)&deviceInput, inputLength * sizeof(uint));
	hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(uint));
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// Copying memory to the GPU
	hipMemcpy(deviceInput, hostInput, inputLength * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(deviceBins, hostBins, NUM_BINS * sizeof(uint), hipMemcpyHostToDevice);
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	// ----------------------------------------------------------

	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");
	// Kernel computation
	compute<<<CEIL(inputLength, BLOCK_SIZE), BLOCK_SIZE>>>(deviceInput, deviceBins, inputLength);
	
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// Copying the GPU memory back to the CPU
	hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(uint), hipMemcpyDeviceToHost);
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// Freeing the GPU memory
	hipFree(deviceBins);
	hipFree(deviceInput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	// -----------------------------------------------------
	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);
}

