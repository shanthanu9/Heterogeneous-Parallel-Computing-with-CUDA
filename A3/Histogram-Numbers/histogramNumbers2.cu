#include "hip/hip_runtime.h"
#include "wb.h"
#include <bits/stdc++.h>
using namespace std;

#define BLOCK_SIZE 1024
#define NUM_BINS 4096
#define BIN_CAP 127
#define CEIL(a, b) ((a-1)/b +1)

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

__global__ void compute(unsigned int * deviceInput, unsigned int * deviceBins) {
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int index = deviceInput[i];

	atomicAdd(&deviceBins[index], 1);
}

__global__ void clean_up(unsigned int * deviceBins) {
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	atomicMin(&deviceBins[i], BIN_CAP);

}
	
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {

	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
		file, line);
		if (abort)
			exit(code);
	}
}


int main(int argc, char *argv[]) {

	int inputLength;
	uint *hostInput;
	uint *hostBins;
	uint *deviceInput;
	uint *deviceBins;

	/* Read input arguments here */
	wbArg_t args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (uint *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostBins = (uint *)calloc(NUM_BINS, sizeof(uint));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating GPU memory.");
	// Allocating GPU memory
	hipMalloc((void **)&deviceInput, inputLength * sizeof(uint));
	hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(uint));
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// Copying memory to the GPU
	CUDA_CHECK(hipDeviceSynchronize());
	hipMemcpy(deviceInput, hostInput, inputLength * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(deviceBins, hostBins, NUM_BINS * sizeof(uint), hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	
	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");
	// Kernel computation

	compute<<<CEIL(inputLength, BLOCK_SIZE), BLOCK_SIZE>>>(deviceInput, deviceBins);
	clean_up<<<CEIL(NUM_BINS, BLOCK_SIZE), BLOCK_SIZE>>>(deviceBins);

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// Copying the GPU memory back to the CPU
	CUDA_CHECK(hipDeviceSynchronize());
	hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(uint), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// Freeing the GPU memory
	hipFree(deviceBins);
	hipFree(deviceInput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	// -----------------------------------------------------
	wbSolution(args, hostBins, NUM_BINS);
	
	free(hostBins);
	free(hostInput);

	return 0;
}

