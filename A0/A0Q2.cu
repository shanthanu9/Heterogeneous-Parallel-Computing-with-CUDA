/*
 *To find sum of two large matrices
 *Compute the speed up obtained by GPU 
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

const int ROW_SIZE = 300, COL_SIZE = 400;
const int MATRIX_BYTES = ROW_SIZE * COL_SIZE * sizeof(int);

//kernal
__global__ void sum(int d_sum[][ROW_SIZE], int d_a[][ROW_SIZE], int d_b[][ROW_SIZE]) {
        d_sum[blockIdx.x][threadIdx.x] = d_a[blockIdx.x][threadIdx.x] + d_b[blockIdx.x][threadIdx.x];
}

//to check the final result
int checkSum(int h_a[][ROW_SIZE], int h_b[][ROW_SIZE], int h_sum[][ROW_SIZE]) {
        int flag = 1;

    for(int i = 0; i < COL_SIZE; i++) {
        for(int j = 0; j < ROW_SIZE; j++) {
           if(h_sum[i][j] != h_a[i][j] + h_b[i][j]) {
                flag = 0;
                break;
           }
        }
    }

    return flag;
}

int main() {
    int h_a[COL_SIZE][ROW_SIZE], h_b[COL_SIZE][ROW_SIZE], h_sum[COL_SIZE][ROW_SIZE];

    for(int i = 0; i < COL_SIZE; i++) {
        for(int j = 0; j < ROW_SIZE; j++) {
           h_a[i][j] = ((int)rand())%1000;
           h_b[i][j] = ((int)rand())%1000;
        }
    }

    int (*d_a)[ROW_SIZE], (*d_b)[ROW_SIZE], (*d_sum)[ROW_SIZE];

    hipMalloc((void**) &d_a, MATRIX_BYTES);
    hipMalloc((void**) &d_b, MATRIX_BYTES);
    hipMalloc((void**) &d_sum, MATRIX_BYTES);

    hipMemcpy(d_a, h_a, MATRIX_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, MATRIX_BYTES, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    sum<<<COL_SIZE, ROW_SIZE>>>(d_sum, d_a, d_b);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(h_sum, d_sum, MATRIX_BYTES, hipMemcpyDeviceToHost);

    if(checkSum(h_a, h_b, h_sum)) {
        printf("The result is computed successfully!\n");
        
        hipEventElapsedTime(&time, start, stop);
        printf("Computation time taken by device: %f\n", time);

        hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);

        for(int i = 0; i < COL_SIZE; i++) {
            for(int j = 0; j < ROW_SIZE; j++) {
                h_sum[i][j] = h_a[i][j] + h_b[i][j];
            }
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&time, start, stop);
        printf("Computation time taken by host: %f\n", time);
    }
    else {
        printf("The result is not computed correctly!");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_sum);

    return 0;
}
